#include "hip/hip_runtime.h"
#include "Input.h"
#include <vector>
#include <iostream>

Input::Input(){
    for(int round = 0; round < nRounds;  round++){
        int i = 0;
        for(int teams = 0; teams < nTeams; teams++) {
            int opp = opponents[round][teams];
            if (opp > 0) {
                games[round][i][0] = teams;
                games[round][i][1] = opp - 1;
                i++;
            }
        }
    }
}


__device__ int Input::getDist(int i, int j)const{
    return dist[i][j];
}

__device__ int Input::getGame(int round, int gameNr, bool away)const{
    return games[round][gameNr][(int)away];
}

__device__ int Input::getOpponent(int team, int round)const{
    return opponents [round][team];
}
