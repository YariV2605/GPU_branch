#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <vector>
#include <algorithm>
#include <array>

#include "Input.h"
#include "ReturnType.h"




//TODO op het einde ook checken of ump op alle locaties geweest is
// bool testFeasibility(int currentPath[nRounds][2], int pathSize, int gameIndex, Input const*const in){
// //q1:
//     for (int q = 0; q < q1 - 1; q++){
//         int gameIndexToCompare = pathSize - 1 - q;
//         if(in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][0]){
//             return false;
//         }
//     }
// //q2:
//     for (int q = 0; q < q2 - 1; q++){
//         int gameIndexToCompare = pathSize - 1 - q;
//         if(
//                 in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][0] ||
//                 in->getGame(pathSize, gameIndex, true ) == currentPath[gameIndexToCompare][0] ||
//                 in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][1] ||
//                 in->getGame(pathSize, gameIndex, true ) == currentPath[gameIndexToCompare][1]
//         ){
//             return false;
//         }
//     }
//     return true;
// }

__device__ bool q1_constr(const ReturnType *const toTest){
    int locationToTest = toTest->getLocation();
    ReturnType* testing = toTest->getPrevious();
    
    for (int q = 0; q < q1 && testing != nullptr; q++){
        if (testing->getLocation() == locationToTest) return false;
        testing = testing->getPrevious();
    }
    return true;
}

__device__ bool q2_constr(const ReturnType *const toTest, const Input in){
    int loc = toTest->getLocation();
    int opponent = in.getOpponent(loc, toTest->getDepth());
    ReturnType* testing = toTest->getPrevious();

    for (int q = 0; q < q2 && testing != nullptr; q++){
        int loc_this = testing->getLocation();
        int opponent_this = in.getOpponent(loc_this, toTest->getDepth());
        if (loc_this == loc             ||
            loc_this == opponent        ||
            opponent_this == loc        ||
            opponent_this == opponent
        ) {
            return false;
        }
        testing = testing->getPrevious();
    }

    return true;
}

__device__ bool beenEverywhere(ReturnType* toTest) {
    if (toTest->getDepth() == nRounds - 1){
        ReturnType* testing = new ReturnType(toTest);
        int teamVisited[nTeams];
        for (int i = 0; i < nTeams; i++){
            teamVisited[i] = 0;
        }
        for (int i = 0; i < nRounds; i++){
            teamVisited[testing->getLocation()] += 1;
            
            testing = testing->getPrevious();
        }
        //verschillend als 0 gevonden is --> er is een team niet bezocht.
        for (int i = 0; i < nTeams; i++){
            if (teamVisited[i] == 0){
                return false;
            }
        }
        return true;
    }
    return true;
}

//WERKT NIET!!!!!
// void DFS_new(const Input *const in, ReturnType** ret, const double v, const double w[nTeams][nRounds]){
//     if (in == nullptr       ||
//         ret == nullptr      ||
//         !q1_constr(*ret)     ||
//         !q2_constr(*ret, in)
//     ) {
//         ret = nullptr;
//         return;
//     }

//     //nRounds - 1 omdat na de laatste ronde niet dieper moeet gegaan worden
//     if ((*ret)->getDepth() < nRounds - 1){
//         //dieper door gaan
//         ReturnType* fromDeeper[nTeams/2];
//         for (int i = 0; i < nTeams/2; i++){
//             int next_loc = in->getGame((*ret)->getDepth()+1, i, false);
//             if(ret == nullptr) std::cout << "weeral" << std:: endl;
//             fromDeeper[i] = new ReturnType(*ret, in->getDist((*ret)->getLocation(), next_loc), next_loc);
//             DFS_new(in, &fromDeeper[i], v, w);
//         }
//         int minDist= 0x7fffffff;
//         *ret = nullptr;
//         for (ReturnType* test: fromDeeper){
//             //FIXME een probleem met get Distance die wordt uitgevoerd op ongeldig memory ?? opgelost door een copyConstructor ??
//             if (test!=nullptr &&
//                 test->getDistance() < minDist &&
//                 beenEverywhere(test)
//             ){
//                 *ret = test;
//                 minDist = test->getDistance();
//             }
//         }
//     }
// }



__global__ void DFS_GPU(const Input in, ReturnType** ret/*, const double v, const double w[nTeams][nRounds]*/){
    int index = threadIdx.x;


    //if infesible (or incorrect call)
    if (ret == nullptr              ||
        !q1_constr(ret[index])      ||
        !q2_constr(ret[index], in)
    ){
        delete (ret[index]);
        ret[index] = nullptr;
        printf("not branching futher\n");
        __syncthreads();
        return;
    }

    printf("depth: %d\n", ret[index]->getDepth());

    //if not yet at deepest level
    if (ret[index]->getDepth() < nRounds - 1){
        //dynamic allocation in order to be able to pass it to the next level (can't be in local)
        ReturnType** nextNodes = (ReturnType**)malloc(nTeams/2 * sizeof(ReturnType*));

        for (int i = 0; i < nTeams/2; i++){
            int nextLocation = in.getGame(ret[index]->getDepth()+1, i, false);
            nextNodes[i] = new ReturnType(ret[index], in.getDist(ret[index]->getLocation(), nextLocation), nextLocation);
        }

        DFS_GPU<<<1, nTeams/2>>>(in, nextNodes/*, v, w*/);
        
        //set this ret to the best possible (or nullptr if none are possible)
        int minDistance = 0x7fffffff;
        ret[index] = nullptr;
        //wait for results from next level before continuing
        hipDeviceSynchronize();
        for (int i = 0; i < nTeams/2; i++){
            if (nextNodes[i] != nullptr &&
                nextNodes[i]->getDistance() < minDistance &&
                // nextNodes[i]->getDepth() == nRounds - 1 &&
                beenEverywhere(nextNodes[i])
            ){
                minDistance = nextNodes[i]->getDistance();
                ret[index] = nextNodes[i];
            }
        }
        //delete all unneeded nodes
        for (int i = 0; i < nTeams/2; i++){
            if(nextNodes[i] != ret[index] /*&& nextNodes[i] != nullptr*/){
                delete nextNodes[i];
                // nextNodes[i] = nullptr;
            }
        }
        free (nextNodes);
    }
    //reached last node
    if (ret[index]->getDepth() == nRounds - 1){
        if(!beenEverywhere(ret[index])) {
            delete (ret[index]);
            ret[index] = nullptr;
            __syncthreads();
            return;
        }
    }
    __syncthreads();
}


__global__ void DFS_GPU2(const Input in, int** ret/*, const double v, const double w[nTeams][nRounds]*/){
    int index = threadIdx.x;


    //if infesible (or incorrect call)
    if (ret == nullptr              /*||
        !q1_constr(ret[index])      ||
        !q2_constr(ret[index], in)*/
    ){
        delete (ret[index]);
        ret[index] = nullptr;
        printf("not branching futher\n");
        __syncthreads();
        return;
    }

    // printf("depth: %d\n", ret[index]->getDepth());

    //if not yet at deepest level
    // if (ret[index]->getDepth() < nRounds - 1){
        //dynamic allocation in order to be able to pass it to the next level (can't be in local)
        int** nextNodes = (int**)malloc(nTeams/2 * sizeof(int*));

        for (int i = 0; i < nTeams/2; i++){
            // int nextLocation = in.getGame(ret[index]->getDepth()+1, i, false);
            nextNodes[i] = new int(*ret[index] + 1);
        }

        DFS_GPU2<<<1, nTeams/2>>>(in, nextNodes/*, v, w*/);
        
        //set this ret to the best possible (or nullptr if none are possible)
        int minDistance = 0x7fffffff;
        ret[index] = nullptr;
        //wait for results from next level before continuing
        hipDeviceSynchronize();
        for (int i = 0; i < nTeams/2; i++){
            if (nextNodes[i] != nullptr //&&
                // nextNodes[i]->getDistance() < minDistance &&
                // nextNodes[i]->getDepth() == nRounds - 1 &&
                // beenEverywhere(nextNodes[i])
            ){
                // minDistance = nextNodes[i]->getDistance();
                ret[index] = nextNodes[i];
            }
        }
        //delete all unneeded nodes
        for (int i = 0; i < nTeams/2; i++){
            if(nextNodes[i] != ret[index] /*&& nextNodes[i] != nullptr*/){
                delete nextNodes[i];
                // nextNodes[i] = nullptr;
            }
        }
        free (nextNodes);
    // }
    //reached last node
    if (*ret[index] == nRounds - 1){
        // if(!beenEverywhere(ret[index])) {
        //     delete (ret[index]);
        //     ret[index] = nullptr;
            __syncthreads();
            return;
        // }
    }
    __syncthreads();
}


__global__ void test(int* i){
    if(*i < 14){
        printf("%d\n", *i);
        int* j = new int(*i + 1);
        test<<<1, 2>>>(j);
    }
}


int main(){
    // int* i_h = new int(0);
    // int* i;
    // hipMalloc(&i, sizeof(int));
    // hipMemcpy(i, i_h, sizeof(int), hipMemcpyHostToDevice);
    // test<<<1, 1>>>(i);
    // hipDeviceSynchronize();
    // std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    // int * t_element_i = new int(0);

    // int** tmp_i = (int**) malloc(sizeof(tmp_i[0]));
    // hipMalloc(&tmp_i[0], sizeof(tmp_i[0][0]));

    // int** t_gpu_i = 0;
    // hipMalloc(&t_gpu_i, sizeof(t_gpu_i[0]));
    
    // hipMemcpy(t_gpu_i, tmp_i, sizeof(t_gpu_i[0]), hipMemcpyHostToDevice);
    // hipMemcpy(tmp_i[0], t_element_i, sizeof(t_gpu_i[0][0]), hipMemcpyHostToDevice);
    

    Input in = Input();

    // DFS_GPU2<<<1, 1>>>(in, t_gpu_i);
    // hipDeviceSynchronize();
    // std::cout << "error: "<< hipGetErrorString(hipGetLastError()) << std::endl;
    double w[nTeams][nRounds];
    for (int i = 0; i < nTeams; i++){
        for (int r = 0; r < nRounds; r++){
            w[i][r] = 0;
        }
    }
    ReturnType* t_element = new ReturnType(3);

    ReturnType** tmp = (ReturnType**) malloc(sizeof(tmp[0]));
    hipMalloc(&tmp[0], sizeof(tmp[0][0]));

    ReturnType** t_gpu = 0;
    hipMalloc(&t_gpu, sizeof(t_gpu[0]));
    
    hipMemcpy(t_gpu, tmp, sizeof(t_gpu[0]), hipMemcpyHostToDevice);
    hipMemcpy(tmp[0], t_element, sizeof(t_gpu[0][0]), hipMemcpyHostToDevice);
    
    free(tmp);



    DFS_GPU<<<1, 1>>>(in, t_gpu/*, 0, w*/);
    hipDeviceSynchronize();
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

    //TODO copy mem to CPU

    ReturnType* a = t_element;
    std::cout << "dist: " << t_element->getDistance() << std::endl;
    std::cout << t_element->getLocation() << " ";
    while (t_element->getPrevious() != nullptr){
        t_element = t_element->getPrevious();
        std::cout << t_element->getLocation() << " ";
    }
    std::cout << std::endl;
    delete(a);
}