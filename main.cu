#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <vector>
#include <algorithm>
#include <array>

#include "Input.h"
#include "ReturnType.h"

#define BLOCKS 1
#define THREAD_PER_BLOCK 1


__global__ void treeSearch(Input* in, int a_s[nTeams][nRounds], double costReduction, double dualCostW[nTeams][nRounds]){
    //TODO
    if(in != nullptr){
        
    }
}

//TODO op het einde ook checken of ump op alle locaties geweest is
bool testFeasibility(int currentPath[nRounds][2], int pathSize, int gameIndex, Input const*const in){
//q1:
    for (int q = 0; q < q1 - 1; q++){
        int gameIndexToCompare = pathSize - 1 - q;
        if(in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][0]){
            return false;
        }
    }
//q2:
    for (int q = 0; q < q2 - 1; q++){
        int gameIndexToCompare = pathSize - 1 - q;
        if(
                in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][0] ||
                in->getGame(pathSize, gameIndex, true ) == currentPath[gameIndexToCompare][0] ||
                in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][1] ||
                in->getGame(pathSize, gameIndex, true ) == currentPath[gameIndexToCompare][1]
        ){
            return false;
        }
    }
    return true;
}


void DFS(Input const*const in, int visited[nRounds][2], int amountVisited, double cost, double w[nTeams][nRounds]){
    if (in == nullptr) return;
    if (amountVisited > 0){
        int round = amountVisited - 1;
        cost += w[visited[round][0]][round];
    }
    if (amountVisited < nRounds){
        for (int game = 0; game < nUmpires; game++){
            if (testFeasibility(visited, amountVisited, game, in)){
                DFS(in, visited, amountVisited, cost, w);
            }
        }
    }
}

bool q1_constr(const ReturnType *const toTest){
    if (toTest == nullptr) std::cout << "nog nullptr's q1" << std::endl;
    int locationToTest = toTest->getLocation();
    ReturnType* testing = toTest->getPrevious();
    
    for (int q = 0; q < q1 && testing != nullptr; q++){
        if(testing == nullptr){
            std::cout << "een nullptr" << std::endl;
        }
        if (testing->getLocation() == locationToTest) return false;
        testing = testing->getPrevious();
    }
    return true;
}

bool q2_constr(const ReturnType *const toTest, const Input *const in){
    if (toTest == nullptr) std::cout << "nog nullptr's q2" << std::endl;
    int loc = toTest->getLocation();
    int opponent = in->getOpponent(loc, toTest->getDepth());
    ReturnType* testing = toTest->getPrevious();

    for (int q = 0; q < q2 && testing != nullptr; q++){
        if(testing == nullptr){
            std::cout << "nullptr" << std::endl;
        }
        int loc_this = testing->getLocation();
        int opponent_this = in->getOpponent(loc_this, toTest->getDepth());
        if (loc_this == loc             ||
            loc_this == opponent        ||
            opponent_this == loc        ||
            opponent_this == opponent
        ) {
            return false;
        }
        testing = testing->getPrevious();
    }

    return true;
}

//TODO een cost calc toevoegen
void DFS_new(const Input *const in, ReturnType* ret, const double v, const double w[nTeams][nRounds]){
    if (in == nullptr       ||
        ret == nullptr      ||
        !q1_constr(ret)     ||
        !q2_constr(ret, in)
    ) {
        ret = nullptr;
        return;
    }

    //nRounds - 1 omdat na de laatste ronde niet dieper moeet gegaan worden
    if (ret->getDepth() < nRounds - 1){
        //dieper door gaan
        ReturnType* fromDeeper[10];
        for (int i = 0; i < nTeams/2; i++){
            int next_loc = in->getGame(ret->getDepth()+1, i, false);
            if(ret == nullptr) std::cout << "weeral" << std:: endl;
            fromDeeper[i] = new ReturnType(ret, in->getDist(ret->getLocation(), next_loc), next_loc);
            DFS_new(in, fromDeeper[i], v, w);
        }
        int minDist= 0x7fffffff;
        ret = nullptr;
        for (ReturnType* test: fromDeeper){
            if (test!=nullptr &&
                test->getDistance() < minDist
            ){
                ret = test;
                minDist = test->getDistance();
            }
        }
    }
    //TODO check of elk team bezocht is --> zoniet ret == nullptr
    if (ret->getDepth() == nRounds - 1){
        ReturnType* testing = ret;
        int teamVisited[nTeams];
        for (int i = 0; i < nTeams; i++){
            teamVisited[i] = 0;
        }
        for (int i = 0; i < nRounds; i++){
            if(testing == nullptr){
                std::cout << "testing is a nullptr" << std::endl;
            }
            teamVisited[testing->getLocation()] += 1;
            
            testing = testing->getPrevious();
        }
        //verschillend als 0 gevonden is --> er is een team niet bezocht.
        if (std::end(teamVisited) != std::find(std::begin(teamVisited), std::end(teamVisited), 0)){
            ret = nullptr;
        }
    }
}


int main(){
    std::cout << "hello";
    Input* i = new Input();
    int a_s[nTeams][nRounds];
    for (int i = 0; i < nTeams; i++){
        for (int r = 0; r < nRounds; r++){
            a_s[i][r] = 0;
        }
    }
    double w[nTeams][nRounds];
    for (int i = 0; i < nTeams; i++){
        for (int r = 0; r < nRounds; r++){
            w[i][r] = 0;
        }
    }
    // treeSearch<<<BLOCKS, THREAD_PER_BLOCK>>>(i, a_s, 0, w);
    int visited[nRounds][2];
    for (int r = 0; r < nRounds; r++){
        visited[r][0] = -1;
        visited[r][1] = -1;
    }
    ReturnType* t = new ReturnType(3);
    DFS_new(i, t, 0, w);
    std::cout << "dist: " << t->getDistance() << std::endl;
    delete(t);
}


//GPU-only function: __device__