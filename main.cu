#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <vector>
#include <algorithm>
#include <array>

#include "Input.h"
#include "ReturnType.h"

#define BLOCKS 1
#define THREAD_PER_BLOCK 1




//TODO op het einde ook checken of ump op alle locaties geweest is
// bool testFeasibility(int currentPath[nRounds][2], int pathSize, int gameIndex, Input const*const in){
// //q1:
//     for (int q = 0; q < q1 - 1; q++){
//         int gameIndexToCompare = pathSize - 1 - q;
//         if(in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][0]){
//             return false;
//         }
//     }
// //q2:
//     for (int q = 0; q < q2 - 1; q++){
//         int gameIndexToCompare = pathSize - 1 - q;
//         if(
//                 in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][0] ||
//                 in->getGame(pathSize, gameIndex, true ) == currentPath[gameIndexToCompare][0] ||
//                 in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][1] ||
//                 in->getGame(pathSize, gameIndex, true ) == currentPath[gameIndexToCompare][1]
//         ){
//             return false;
//         }
//     }
//     return true;
// }

__device__ bool q1_constr(const ReturnType *const toTest){
    int locationToTest = toTest->getLocation();
    ReturnType* testing = toTest->getPrevious();
    
    for (int q = 0; q < q1 && testing != nullptr; q++){
        if (testing->getLocation() == locationToTest) return false;
        testing = testing->getPrevious();
    }
    return true;
}

__device__ bool q2_constr(const ReturnType *const toTest, const Input in){
    int loc = toTest->getLocation();
    int opponent = in.getOpponent(loc, toTest->getDepth());
    ReturnType* testing = toTest->getPrevious();

    for (int q = 0; q < q2 && testing != nullptr; q++){
        int loc_this = testing->getLocation();
        int opponent_this = in.getOpponent(loc_this, toTest->getDepth());
        if (loc_this == loc             ||
            loc_this == opponent        ||
            opponent_this == loc        ||
            opponent_this == opponent
        ) {
            return false;
        }
        testing = testing->getPrevious();
    }

    return true;
}

__device__ bool beenEverywhere(ReturnType* toTest) {
    if (toTest->getDepth() == nRounds - 1){
        ReturnType* testing = new ReturnType(toTest);
        int teamVisited[nTeams];
        for (int i = 0; i < nTeams; i++){
            teamVisited[i] = 0;
        }
        for (int i = 0; i < nRounds; i++){
            teamVisited[testing->getLocation()] += 1;
            
            testing = testing->getPrevious();
        }
        //verschillend als 0 gevonden is --> er is een team niet bezocht.
        for (int i = 0; i < nTeams; i++){
            if (teamVisited[i] == 0){
                return false;
            }
        }
        return true;
    }
    return true;
}

//WERKT NIET!!!!!
// void DFS_new(const Input *const in, ReturnType** ret, const double v, const double w[nTeams][nRounds]){
//     if (in == nullptr       ||
//         ret == nullptr      ||
//         !q1_constr(*ret)     ||
//         !q2_constr(*ret, in)
//     ) {
//         ret = nullptr;
//         return;
//     }

//     //nRounds - 1 omdat na de laatste ronde niet dieper moeet gegaan worden
//     if ((*ret)->getDepth() < nRounds - 1){
//         //dieper door gaan
//         ReturnType* fromDeeper[nTeams/2];
//         for (int i = 0; i < nTeams/2; i++){
//             int next_loc = in->getGame((*ret)->getDepth()+1, i, false);
//             if(ret == nullptr) std::cout << "weeral" << std:: endl;
//             fromDeeper[i] = new ReturnType(*ret, in->getDist((*ret)->getLocation(), next_loc), next_loc);
//             DFS_new(in, &fromDeeper[i], v, w);
//         }
//         int minDist= 0x7fffffff;
//         *ret = nullptr;
//         for (ReturnType* test: fromDeeper){
//             //FIXME een probleem met get Distance die wordt uitgevoerd op ongeldig memory ?? opgelost door een copyConstructor ??
//             if (test!=nullptr &&
//                 test->getDistance() < minDist &&
//                 beenEverywhere(test)
//             ){
//                 *ret = test;
//                 minDist = test->getDistance();
//             }
//         }
//     }
// }



__global__ void DFS_GPU(const Input in, ReturnType** ret, const double v, const double w[nTeams][nRounds]){
    //FIXME ret[0] is een illegal memory acces, cuda malloc zal dus niet werken zoals ik bedoeld had.
    printf("%d", ret[0]);
    int index = threadIdx.x;


    //if infesible (or incorrect call)
    if (ret == nullptr              ||
        !q1_constr(ret[index])      ||
        !q2_constr(ret[index], in)
    ){
        delete (ret[index]);
        ret[index] = nullptr;
        __syncthreads();
        return;
    }

    printf("depth: %d", ret[index]->getDepth());

    //if not yet at deepest level
    if (ret[index]->getDepth() < nRounds - 1){
        ReturnType* nextNodes[nTeams/2];
        ReturnType** nextNodes_gpu = nullptr;

        for (int i = 0; i < nTeams/2; i++){
            int nextLocation = in.getGame(ret[index]->getDepth()+1, i, false);
            nextNodes[i] = new ReturnType(ret[index], in.getDist(ret[index]->getLocation(), nextLocation), nextLocation);
        }

        DFS_GPU<<<1, nTeams/2>>>(in, nextNodes_gpu, v, w);
        
        //set this ret to the best possible (or nullptr if none are possible)
        int minDistance = 0x7fffffff;
        ret[index] = nullptr;
        //wait for results from next level before continuing
        hipDeviceSynchronize();
        for (int i = 0; i < nTeams/2; i++){
            if (nextNodes[i] != nullptr &&
                nextNodes[i]->getDistance() < minDistance &&
                beenEverywhere(nextNodes[i])
            ){
                minDistance = nextNodes[i]->getDistance();
                ret[index] = nextNodes[i];
            }
        }
        //delete all unneeded nodes
        for (int i = 0; i < nTeams/2; i++){
            if(nextNodes[i] != ret[index]){
                delete nextNodes[i];
            }
        }
    }
    //reached last node
    if (ret[index]->getDepth() == nRounds - 1){
        if(!beenEverywhere(ret[index])) {
            delete (ret[index]);
            ret[index] = nullptr;
            __syncthreads();
            return;
        }
    }
    __syncthreads();
}



int main(){
    Input in = Input();
    double w[nTeams][nRounds];
    for (int i = 0; i < nTeams; i++){
        for (int r = 0; r < nRounds; r++){
            w[i][r] = 0;
        }
    }
    ReturnType* t_element = new ReturnType(3);
    ReturnType* t_element_gpu;
    ReturnType** t_gpu;


    //FIXME
    hipMalloc(&t_element_gpu, 1*sizeof(ReturnType));//deze array zal maar 1 element groot zijn
    hipMemcpy(t_element_gpu, t_element, sizeof(ReturnType), hipMemcpyHostToDevice);

    hipMalloc(&t_gpu, 1*sizeof(ReturnType*));
    hipMemcpy(t_gpu, &t_element, sizeof(ReturnType*), hipMemcpyHostToDevice);



    DFS_GPU<<<1, 1>>>(in, &t_element_gpu, 0, w);
    hipDeviceSynchronize();
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

    hipMemcpy(t_element, t_element_gpu, sizeof(ReturnType), hipMemcpyDeviceToHost);

    ReturnType* a = t_element;
    std::cout << "dist: " << t_element->getDistance() << std::endl;
    std::cout << t_element->getLocation() << " ";
    while (t_element->getPrevious() != nullptr){
        t_element = t_element->getPrevious();
        std::cout << t_element->getLocation() << " ";
    }
    std::cout << std::endl;
    delete(a);
}