#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <vector>
#include <algorithm>
#include <array>

#include "Input.h"
#include "ReturnType.h"



__device__ bool q1_constr(const ReturnType *const toTest){
    int locationToTest = toTest->getLocation();
    ReturnType* testing = toTest->getPrevious();
    
    for (int q = 0; q < q1 && testing != nullptr; q++){
        if (testing->getLocation() == locationToTest) return false;
        testing = testing->getPrevious();
    }
    return true;
}

__device__ bool q2_constr(const ReturnType *const toTest, const Input in){
    int loc = toTest->getLocation();
    int opponent = in.getOpponent(loc, toTest->getDepth());
    ReturnType* testing = toTest->getPrevious();

    for (int q = 0; q < q2 && testing != nullptr; q++){
        int loc_this = testing->getLocation();
        int opponent_this = in.getOpponent(loc_this, toTest->getDepth());
        if (loc_this == loc             ||
            loc_this == opponent        ||
            opponent_this == loc        ||
            opponent_this == opponent
        ) {
            return false;
        }
        testing = testing->getPrevious();
    }

    return true;
}

__device__ bool beenEverywhere(ReturnType* toTest) {
    if (toTest->getDepth() == nRounds - 1){
        ReturnType* testing = new ReturnType(toTest);
        int teamVisited[nTeams];
        for (int i = 0; i < nTeams; i++){
            teamVisited[i] = 0;
        }
        for (int i = 0; i < nRounds; i++){
            teamVisited[testing->getLocation()] += 1;
            
            testing = testing->getPrevious();
        }
        //verschillend als 0 gevonden is --> er is een team niet bezocht.
        for (int i = 0; i < nTeams; i++){
            if (teamVisited[i] == 0){
                return false;
            }
        }
        return true;
    }
    return true;
}



__global__ void DFS_GPU(const Input in, ReturnType** ret/*, const double v, const double w[nTeams][nRounds]*/){
    int index = threadIdx.x;


    //if infeasible (or incorrect call)
    if (ret == nullptr              ||
        !q1_constr(ret[index])      ||
        !q2_constr(ret[index], in)
    ){
        delete (ret[index]);
        ret[index] = nullptr;
        printf("not branching futher\n");
        __syncthreads();
        return;
    }

    printf("depth: %d\n", ret[index]->getDepth());

    //if not yet at deepest level
    if (ret[index]->getDepth() < nRounds - 1){
        //dynamic allocation in order to be able to pass it to the next level (can't be in local)
        ReturnType** nextNodes = (ReturnType**)malloc(nTeams/2 * sizeof(ReturnType*));

        for (int i = 0; i < nTeams/2; i++){
            int nextLocation = in.getGame(ret[index]->getDepth()+1, i, false);
            nextNodes[i] = new ReturnType(ret[index], in.getDist(ret[index]->getLocation(), nextLocation), nextLocation);
        }

        DFS_GPU<<<1, nTeams/2>>>(in, nextNodes/*, v, w*/);
        
        //set this ret to the best possible (or nullptr if none are possible)
        int minDistance = 0x7fffffff;
        ret[index] = nullptr;
        //wait for results from next level before continuing
        hipDeviceSynchronize();
        for (int i = 0; i < nTeams/2; i++){
            if (nextNodes[i] != nullptr &&
                nextNodes[i]->getDistance() < minDistance &&
                // nextNodes[i]->getDepth() == nRounds - 1 &&
                beenEverywhere(nextNodes[i])
            ){
                minDistance = nextNodes[i]->getDistance();
                ret[index] = nextNodes[i];
            }
        }
        //delete all unneeded nodes
        for (int i = 0; i < nTeams/2; i++){
            if(nextNodes[i] != ret[index] /*&& nextNodes[i] != nullptr*/){
                delete nextNodes[i];
                // nextNodes[i] = nullptr;
            }
        }
        free (nextNodes);
    }
    //reached last node
    if (ret[index]->getDepth() == nRounds - 1){
        if(!beenEverywhere(ret[index])) {
            delete (ret[index]);
            ret[index] = nullptr;
            __syncthreads();
            return;
        }
    }
    __syncthreads();
}



__global__ void test(int* i){
    if(*i < 14){
        printf("%d\n", *i);
        int* j = new int(*i + 1);
        test<<<1, 2>>>(j);
    }
}


int main(){
    Input in = Input();
    double w[nTeams][nRounds];
    for (int i = 0; i < nTeams; i++){
        for (int r = 0; r < nRounds; r++){
            w[i][r] = 0;
        }
    }
    ReturnType* t_element = new ReturnType(3);

    ReturnType** tmp = (ReturnType**) malloc(sizeof(tmp[0]));
    hipMalloc(&tmp[0], sizeof(tmp[0][0]));

    ReturnType** t_gpu = 0;
    hipMalloc(&t_gpu, sizeof(t_gpu[0]));
    
    hipMemcpy(t_gpu, tmp, sizeof(t_gpu[0]), hipMemcpyHostToDevice);
    hipMemcpy(tmp[0], t_element, sizeof(t_gpu[0][0]), hipMemcpyHostToDevice);
    
    free(tmp);



    DFS_GPU<<<1, 1>>>(in, t_gpu/*, 0, w*/);
    hipDeviceSynchronize();
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

    //TODO copy mem to CPU

    ReturnType* a = t_element;
    std::cout << "dist: " << t_element->getDistance() << std::endl;
    std::cout << t_element->getLocation() << " ";
    while (t_element->getPrevious() != nullptr){
        t_element = t_element->getPrevious();
        std::cout << t_element->getLocation() << " ";
    }
    std::cout << std::endl;
    delete(a);
}