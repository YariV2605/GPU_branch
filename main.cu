#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <vector>
#include "Input.h"
#define BLOCKS 1
#define THREAD_PER_BLOCK 1


__global__ void treeSearch(Input* in, int a_s[nTeams][nRounds], double costReduction, double dualCostW[nTeams][nRounds]){
    //TODO
    if(in != nullptr){
        
    }
}

//TODO op het einde ook checken of ump op alle locaties geweest is
bool testFeasibility(int currentPath[nRounds][2], int pathSize, int gameIndex, Input const*const in){
//q1:
    for (int q = 0; q < q1 - 1; q++){
        int gameIndexToCompare = pathSize - 1 - q;
        if(in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][0]){
            return false;
        }
    }
//q2:
    for (int q = 0; q < q2 - 1; q++){
        int gameIndexToCompare = pathSize - 1 - q;
        if(
                in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][0] ||
                in->getGame(pathSize, gameIndex, true ) == currentPath[gameIndexToCompare][0] ||
                in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][1] ||
                in->getGame(pathSize, gameIndex, true ) == currentPath[gameIndexToCompare][1]
        ){
            return false;
        }
    }
    return true;
}


void DFS(Input const*const in, int visited[nRounds][2], int amountVisited, double cost, double w[nTeams][nRounds]){
    if (in == nullptr) return;
    if (amountVisited > 0){
        int round = amountVisited - 1;
        cost += w[visited[round][0]][round];
    }
    if (amountVisited < nRounds){
        for (int game = 0; game < nUmpires; game++){
            if (testFeasibility(visited, amountVisited, game, in)){
                DFS(in, visited, amountVisited, cost, w);
            }
        }
    }
}


int main(){
    std::cout << "hello";
    Input* i = new Input();
    int a_s[nTeams][nRounds];
    for (int i = 0; i < nTeams; i++){
        for (int r = 0; r < nRounds; r++){
            a_s[i][r] = 0;
        }
    }
    double w[nTeams][nRounds];
    for (int i = 0; i < nTeams; i++){
        for (int r = 0; r < nRounds; r++){
            w[i][r] = 0;
        }
    }
    // treeSearch<<<BLOCKS, THREAD_PER_BLOCK>>>(i, a_s, 0, w);
    int visited[nRounds][2];
    for (int r = 0; r < nRounds; r++){
        visited[r][0] = -1;
        visited[r][1] = -1;
    }
    DFS(i, visited, 0, 0, w);
}