#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <vector>
#include <algorithm>
#include <array>

#include "Input.h"
#include "ReturnType.h"

#define BLOCKS 1
#define THREAD_PER_BLOCK 1




//TODO op het einde ook checken of ump op alle locaties geweest is
bool testFeasibility(int currentPath[nRounds][2], int pathSize, int gameIndex, Input const*const in){
//q1:
    for (int q = 0; q < q1 - 1; q++){
        int gameIndexToCompare = pathSize - 1 - q;
        if(in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][0]){
            return false;
        }
    }
//q2:
    for (int q = 0; q < q2 - 1; q++){
        int gameIndexToCompare = pathSize - 1 - q;
        if(
                in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][0] ||
                in->getGame(pathSize, gameIndex, true ) == currentPath[gameIndexToCompare][0] ||
                in->getGame(pathSize, gameIndex, false) == currentPath[gameIndexToCompare][1] ||
                in->getGame(pathSize, gameIndex, true ) == currentPath[gameIndexToCompare][1]
        ){
            return false;
        }
    }
    return true;
}

__device__ bool q1_constr(const ReturnType *const toTest){
    if (toTest == nullptr) std::cout << "nog nullptr's q1" << std::endl;
    int locationToTest = toTest->getLocation();
    ReturnType* testing = toTest->getPrevious();
    
    for (int q = 0; q < q1 && testing != nullptr; q++){
        if(testing == nullptr){
            std::cout << "een nullptr" << std::endl;
        }
        if (testing->getLocation() == locationToTest) return false;
        testing = testing->getPrevious();
    }
    return true;
}

__device__ bool q2_constr(const ReturnType *const toTest, const Input *const in){
    if (toTest == nullptr) std::cout << "nog nullptr's q2" << std::endl;
    int loc = toTest->getLocation();
    int opponent = in->getOpponent(loc, toTest->getDepth());
    ReturnType* testing = toTest->getPrevious();

    for (int q = 0; q < q2 && testing != nullptr; q++){
        if(testing == nullptr){
            std::cout << "nullptr" << std::endl;
        }
        int loc_this = testing->getLocation();
        int opponent_this = in->getOpponent(loc_this, toTest->getDepth());
        if (loc_this == loc             ||
            loc_this == opponent        ||
            opponent_this == loc        ||
            opponent_this == opponent
        ) {
            return false;
        }
        testing = testing->getPrevious();
    }

    return true;
}

__device__ bool beenEverywhere(ReturnType* toTest) {
    if (toTest->getDepth() == nRounds - 1){
        ReturnType* testing = new ReturnType(toTest);
        int teamVisited[nTeams];
        for (int i = 0; i < nTeams; i++){
            teamVisited[i] = 0;
        }
        for (int i = 0; i < nRounds; i++){
            if(testing == nullptr){
                std::cout << "testing is a nullptr" << std::endl;
            }
            teamVisited[testing->getLocation()] += 1;
            
            testing = testing->getPrevious();
        }
        //verschillend als 0 gevonden is --> er is een team niet bezocht.
        if (std::end(teamVisited) != std::find(std::begin(teamVisited), std::end(teamVisited), 0)){
            return false;
        }
    }
    return true;
}

//WERKT NIET!!!!!
// void DFS_new(const Input *const in, ReturnType** ret, const double v, const double w[nTeams][nRounds]){
//     if (in == nullptr       ||
//         ret == nullptr      ||
//         !q1_constr(*ret)     ||
//         !q2_constr(*ret, in)
//     ) {
//         ret = nullptr;
//         return;
//     }

//     //nRounds - 1 omdat na de laatste ronde niet dieper moeet gegaan worden
//     if ((*ret)->getDepth() < nRounds - 1){
//         //dieper door gaan
//         ReturnType* fromDeeper[nTeams/2];
//         for (int i = 0; i < nTeams/2; i++){
//             int next_loc = in->getGame((*ret)->getDepth()+1, i, false);
//             if(ret == nullptr) std::cout << "weeral" << std:: endl;
//             fromDeeper[i] = new ReturnType(*ret, in->getDist((*ret)->getLocation(), next_loc), next_loc);
//             DFS_new(in, &fromDeeper[i], v, w);
//         }
//         int minDist= 0x7fffffff;
//         *ret = nullptr;
//         for (ReturnType* test: fromDeeper){
//             //FIXME een probleem met get Distance die wordt uitgevoerd op ongeldig memory ?? opgelost door een copyConstructor ??
//             if (test!=nullptr &&
//                 test->getDistance() < minDist &&
//                 beenEverywhere(test)
//             ){
//                 *ret = test;
//                 minDist = test->getDistance();
//             }
//         }
//     }
// }



__global__ void DFS_GPU(const Input *const in, ReturnType** ret, const double v, const double w[nTeams][nRounds]){
    int index = threadIdx.x;


    //if infesible (or incorrect call)
    if (in  == nullptr      ||
        ret == nullptr      ||
        !q1_constr(ret[index])     ||
        !q2_constr(ret[index], in)
    ){
        delete (ret[index]);
        ret[index] = nullptr;
        return;
    }

    //if not yet at deepest level
    if (ret[index]->getDepth() < nRounds - 1){
        ReturnType* nextNodes[nTeams/2];
        for (int i = 0; i < nTeams/2; i++){
            int nextLocation = in->getGame(ret[index]->getDepth()+1, i, false);
            nextNodes[i] = new ReturnType(ret[index], in->getDist(ret[index]->getLocation(), nextLocation), nextLocation);
        }
        DFS_GPU<<<1, nTeams/2>>>(in, nextNodes, v, w);
        //set this ret to the best possible (or nullptr if none are possible)
        int minDistance = 0x7fffffff;
        ret[index] = nullptr;
        for (int i = 0; i < nTeams/2; i++){
            if (nextNodes[i] != nullptr &&
                nextNodes[i]->getDistance() < minDistance &&
                beenEverywhere(nextNodes[i])
            ){
                minDistance = nextNodes[i]->getDistance();
                ret[index] = nextNodes[i];
            }
        }
        //delete all unneeded nodes
        for (int i = 0; i < nTeams/2; i++){
            if(nextNodes[i] != ret[index]){
                delete nextNodes[i];
            }
        }
    }
    //reached last node
    if (ret[index]->getDepth() == nRounds - 1){
        if(!beenEverywhere(ret[index])) {
            delete (ret[index]);
            ret[index] = nullptr;
            return;
        }
    }
}



int main(){
    Input* in = new Input();
    double w[nTeams][nRounds];
    for (int i = 0; i < nTeams; i++){
        for (int r = 0; r < nRounds; r++){
            w[i][r] = 0;
        }
    }
    ReturnType* t_element = new ReturnType(3);
    Input* in_gpu;
    ReturnType* t_element_gpu;
    ReturnType** t_gpu;



    hipMalloc(&in_gpu, sizeof(Input));
    hipMemcpy(in_gpu, in, sizeof(Input), hipMemcpyHostToDevice);

    hipMalloc(&t_element_gpu, sizeof(ReturnType));//deze array zal maar 1 element groot zijn
    hipMemcpy(t_element_gpu, t_element, sizeof(ReturnType), hipMemcpyHostToDevice);

    hipMalloc(&t_gpu, 1*sizeof(ReturnType*));
    hipMemcpy(t_gpu, &t_element, sizeof(ReturnType*), hipMemcpyHostToDevice);



    DFS_GPU<<<1, 1>>>(in, &t_element, 0, w);
    //TODO mem terug naar host copy-en

    ReturnType* a = t_element;
    std::cout << "dist: " << t_element->getDistance() << std::endl;
    std::cout << t_element->getLocation() << " ";
    while (t_element->getPrevious() != nullptr){
        t_element = t_element->getPrevious();
        std::cout << t_element->getLocation() << " ";
    }
    std::cout << std::endl;
    delete(a);
}