#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <vector>
#include "Input.h"
#define BLOCKS 1
#define THREAD_PER_BLOCK 1
#define nTeams 20
#define nRounds 38
#define nUmpires 10

__global__ void treeSearch(Input* in, int a_s[nTeams][nRounds], double costReduction, double dualCostW[nTeams][nRounds]){
    //TODO
    if(in != nullptr){
        
    }
}

int main(){
    Input* i = new Input();
    int a_s[nTeams][nRounds];
    for (int i = 0; i < nTeams; i++){
        for (int r = 0; r < nRounds; r++){
            a_s[i][r] = 0;
        }
    }
    int w[nTeams][nRounds];
    for (int i = 0; i < nTeams; i++){
        for (int r = 0; r < nRounds; r++){
            w[i][r] = 0;
        }
    }
    treeSearch<<<BLOCKS, THREAD_PER_BLOCK>>>(i, 0, a_s, 0);
}


void DFS(Input const*const in, std::vector<int[2]> visited, double cost, int[nTeams][nRounds]& w){
    if (in == nullptr) return;
    int round = visited.lenth() - 1
    cost += w[visited[round][0]][round];
    for (int game = 0; game < nUmpires; game++){
        if (testFeasibility()){
            DFS(in, visited, cost, w);
        }
    }
}
//TODO op het einde ook checken of ump op alle locaties geweest is
bool testFeasibility(std::vector<int[2]> currentPath, int gameIndex, Input const*const in){
//q1:
    for (int q = 0; q < q1 - 1; q++){
        if(in->getGame(currentPath.length(), gameIndex, false) == currentPath.get(currentPath.length() - 1 - q)){
            return false;
        }
    }
//q2:
    for (int q = 0; q < q2 - 1; q++){
        int gameIndexToCompare = currentPath.length() - 1 - q;
        if(
                in->getGame(currentPath.length(), gameIndex, false) == currentPath[gameIndexToCompare][0] ||
                in->getGame(currentPath.length(), gameIndex, true ) == currentPath[gameIndexToCompare][0] ||
                in->getGame(currentPath.length(), gameIndex, false) == currentPath[gameIndexToCompare][1] ||
                in->getGame(currentPath.length(), gameIndex, true ) == currentPath[gameIndexToCompare][1]
        ){
            return false;
        }
    }
    return true;
}